#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "demo.cuh"

__global__ void add_kernel(float* A, float* B, float* C)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
    //printf("%d\n", C[idx]);
}

int testCUDA()
{
    int num = 100;
    float a[num];
    float b[num];
    float c[num];
    float a_test[num];
    
    std::vector<float> a_vec;
    std::vector<float> b_vec;
    std::vector<float> c_vec;

    int block_num = 20;
    int block_size = num / block_num;
    hipError_t cudaStatus;

    for (int i = 0; i < num; i++)
    {
        a[i] = i;
        b[i] = i;
        //a_vec.push_back(i);
        //b_vec.push_back(i);
        a_vec.emplace_back(i);
        b_vec.emplace_back(i);
    }

    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, sizeof(float) * num);
    hipMalloc((void **)&b_d, sizeof(float) * num);
    hipMalloc((void **)&c_d, sizeof(float) * num);
 
    //hipMemcpy(a_d, a, num * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(b_d, b, num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(a_d, a_vec.data(), num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_vec.data(), num * sizeof(int), hipMemcpyHostToDevice);

    add_kernel<<<block_num, block_size>>>(a_d, b_d, c_d);
    
    cudaStatus = hipGetLastError();
    if(cudaStatus != hipSuccess)
    {
        std::cout << "add_kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
        goto Error;
    }
    //hipDeviceSynchronize(); 
    
    cudaStatus = hipMemcpy(a_test, a_d, num * sizeof(int), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess)
    {
        std::cout << "cuda memcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
        goto Error;
    }

    for (int i=0; i<num; i++)
    {
        std::cout << i << ": " << *(a_test + i) << std::endl;
    }

    cudaStatus = hipMemcpy(c, c_d, num * sizeof(int), hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess)
    {
        std::cout << "cuda memcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
        goto Error;
    }


    for (int i = 0; i < num; i++)
    {
    	//std::cout << "c[" << i << "]: " << c[i] << std::endl;
    }

Error:
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}
